#include "hip/hip_runtime.h"
/* Part of program for solving linear equations using CUDA C */ 
/* Uses Gaussian elimination */ 
/* Files requirede : Kernel.cu , main.cpp, fread.cpp, common.h , DeviceFunc.cu ( this file), data.txt */

//Assigning memory on device and defining Thread Block size 
// Call to the Kernel(function) that will run on the GPU 

#include<hip/hip_runtime.h> 
#include<stdio.h> 
#include "Common.h" 

__device__ __global__ void Kernel(float *, float * ,int ); 

void DeviceFunc(float *temp_h , int numvar , float *temp1_h) 
{ 
	float *a_d , *b_d; 

	//Memory allocation on the device 
	
	hipMalloc(&a_d,sizeof(float)*(numvar)*(numvar+1)); 
	
	hipMalloc(&b_d,sizeof(float)*(numvar)*(numvar+1)); 
	
	//Copying data to device from host 
	
	hipMemcpy(a_d, temp_h, sizeof(float)*numvar*(numvar+1),cudaMemcpyHostTo Device); 
	
	//Defining size of Thread Block 
	dim3 dimBlock(numvar+1,numvar,1); 
	dim3 dimGrid(1,1,1); 
	
	//Kernel call 
	Kernel<<<dimGrid , dimBlock>>>(a_d , b_d , numvar); 
	
	//Coping data to host from device 
	hipMemcpy(temp1_h,b_d,sizeof(float)*numvar*(numvar+1),cudaMemcpyDeviceT oHost);

	//Deallocating memory on the device 
	hipFree(a_d); 
	hipFree(b_d); 
}


