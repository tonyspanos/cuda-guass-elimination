#include "hip/hip_runtime.h"
/* Part of program for solving linear equations using CUDA C */
/* Uses Gaussian elimination */ 
/* Files requirede : Kernel.cu (this file) , main.cpp, fread.cpp, common.h, Devi ceFunc.cu, data.txt */

//Kernel function that executes on the device 

#include "Common.h" 
#include<hip/hip_runtime.h> 

__device__ __global__ void Kernel(float *a_d , float *b_d ,int size) 
{ 
	int idx = threadIdx.x ;
	int idy = threadIdx.y ; 
	
	//int width = size ; 
	//int height = size ; 
	
	//Allocating memory in the share memory of the device 
	
	__shared__ float temp[16][16]; 
	
	//Copying the data to the shared memory 
	
	temp[idy][idx] = a_d[(idy * (size+1)) + idx] ;
	
	for(int i =1 ; i<size ;i++) 
	{ 
		if((idy + i) < size) // NO Thread divergence here 
		{ 
			float var1 =(-1)*( temp[i-1][i-1]/temp[i+idy][i-1]);
			temp[i+idy][idx] = temp[i-1][idx] +((var1) * (temp[i+idy ][idx]));
		} 
		
		__syncthreads(); //Synchronizing all threads before Next iterat ion 
	} 
	
	b_d[idy*(size+1) + idx] = temp[idy][idx]; 
}

